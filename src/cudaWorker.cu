#include "hip/hip_runtime.h"
#include <cudaWorker.h>
#include <math.h>
#include <processor.h>
#include <iostream>
#include <string>
#include <vector>

__global__ void calculate(int row, char* data, int queryLength, char* query,
                          int* result) {
  int move[8] = {0, 1, 1, 0, 1, 1, -1, -1};

  int col = row;
  int threadCount = blockDim.x * gridDim.x;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int segment = row / threadCount;

  int i = segment * tid;
  int j = 0;

  int iMax = segment * (tid + 1);
  if (tid == (threadCount - 1)) {
    iMax = row;
  }

  bool found = false;

  while (!found && i < iMax) {
    j = 0;
    while (!found && j < col) {
      // If first letter matched
      if (query[0] == data[i * row + j]) {
        // For all direction
        for (int dir = 0; dir < 4; dir++) {
          direction currDir = static_cast<direction>(dir);
          int k = 0;
          while (!found && k < queryLength) {
            int tempRow = i + move[currDir * 2] * k;
            int tempCol = j + move[currDir * 2 + 1] * k;

            if (tempRow >= row || tempCol >= col || tempRow < 0 ||
                tempCol < 0) {
              break;
            }

            if (data[tempRow * row + tempCol] != query[k]) {
              break;
            }

            if (k == queryLength - 1) {
              found = true;
              result[0] = true;
              result[1] = i + 1;
              result[2] = j + 1;
              result[3] = currDir;
            }
            k++;
          }
        }
      }
      j++;
    }
    i++;
  }
}

void docuda(char* cpuData, int row, std::string cpuQuery, int threadCount,
            int* cpuResult) {
  char* serialData;
  char* query;
  int* result;

  hipMallocManaged(&serialData, row * row * sizeof(char));
  hipMallocManaged(&result, 4 * sizeof(int));
  hipMallocManaged(&query, cpuQuery.size() * sizeof(char));

  for (int i = 0; i < cpuQuery.size(); i++) {
    query[i] = cpuQuery[i];
  }

  for (int i = 0; i < row * row; i++) {
    serialData[i] = cpuData[i];
  }

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  std::cout << "Device info" << std::endl;
  std::cout << props.name << ": " << props.major << "." << props.minor
            << std::endl;
  std::cout << "  Warp size            : " << props.warpSize << std::endl;
  std::cout << "  Threads per block    : " << props.maxThreadsPerBlock
            << std::endl;
  std::cout << "  SM (Processor) count : " << props.multiProcessorCount
            << std::endl;

  int threadPerBlock = 128;
  std::cout << "Thread per block (recommended 128/256): ";
  std::cin >> threadPerBlock;

  int blockCount = threadCount / threadPerBlock;

  std::cout << "Using " << blockCount << " blocks" << std::endl;

  calculate<<<blockCount, threadPerBlock>>>(row, serialData, cpuQuery.size(),
                                            query, result);

  hipDeviceSynchronize();

  for (int i = 0; i < 4; i++) {
    // std::cout << result[i] << std::endl;
    cpuResult[i] = result[i];
  }

  hipFree(serialData);
  hipFree(query);
  hipFree(result);
}