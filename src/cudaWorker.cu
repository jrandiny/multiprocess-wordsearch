#include "hip/hip_runtime.h"
#include <cudaWorker.h>
#include <math.h>
#include <processor.h>
#include <iostream>
#include <string>
#include <utility>
#include <vector>

__global__ void calculate(int row, char* data, int queryLength, char* query,
                          int* result) {
  int move[8] = {0, 1, 1, 0, 1, 1, -1, -1};

  int col = row;
  int threadCount = blockDim.x;

  int tid = threadIdx.x;
  int segment = row / threadCount;

  int i = segment * tid;
  int j = 0;

  int iMax = segment * (tid + 1);
  if (tid == (threadCount - 1)) {
    iMax = row;
  }

  bool found = false;

  while (!found && i < iMax) {
    j = 0;
    while (!found && j < col) {
      // If first letter matched
      if (query[0] == data[i * row + j]) {
        // For all direction
        for (int dir = 0; dir < 4; dir++) {
          direction currDir = static_cast<direction>(dir);
          int k = 0;
          while (!found && k < queryLength) {
            int tempRow = i + move[currDir * 2] * k;
            int tempCol = j + move[currDir * 2 + 1] * k;

            if (tempRow >= row || tempCol >= col || tempRow < 0 ||
                tempCol < 0) {
              break;
            }

            if (data[tempRow * row + tempCol] != query[k]) {
              break;
            }

            if (k == queryLength - 1) {
              found = true;
              result[0] = true;
              result[1] = i + 1;
              result[2] = j + 1;
              result[3] = currDir;
            }
            k++;
          }
        }
      }
      j++;
    }
    i++;
  }
}

void docuda(char* cpuData, int row, std::string cpuQuery, int threadCount,
            int* cpuResult) {
  char* serialData;
  char* query;
  int* result;

  hipMallocManaged(&serialData, row * row * sizeof(char));
  hipMallocManaged(&result, 4 * sizeof(int));
  hipMallocManaged(&query, cpuQuery.size() * sizeof(char));

  for (int i = 0; i < cpuQuery.size(); i++) {
    query[i] = cpuQuery[i];
  }

  for (int i = 0; i < row * row; i++) {
    serialData[i] = cpuData[i];
  }

  calculate<<<1, threadCount>>>(row, serialData, cpuQuery.size(), query,
                                result);

  hipDeviceSynchronize();

  for (int i = 0; i < 4; i++) {
    // std::cout << result[i] << std::endl;
    cpuResult[i] = result[i];
  }

  hipFree(serialData);
  hipFree(query);
  hipFree(result);
}